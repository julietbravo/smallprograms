
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <stdlib.h>
#include <cstdio>

__host__ __device__ inline double dg4(const double v1, const double v2, const double v3, const double v4, const double v5, const double v6, const double v7)
{
    return (1./576.)*(v1+v7) + (-54./576.)*(v2+v6) + (783./576.)*(v3+v5) + (-1460./576.)*v4;
}

/*
4th order diffusion (3D), quite similar to CPU implementation MicroHH
*/
void diff_cpu_3d(double * const __restrict__ at, const double * const __restrict__ a,
        const double dxidxi, const double dyidyi, const double dzidzi,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart, const int kend,
        const int icells, const int ijcells)
{
    const int ii1 = 1;
    const int ii2 = 2;
    const int ii3 = 3;
    const int jj1 = 1*icells;
    const int jj2 = 2*icells;
    const int jj3 = 3*icells;
    const int kk1 = 1*ijcells;
    const int kk2 = 2*ijcells;
    const int kk3 = 3*ijcells;

    const double visc = 0.1;

    for (int k=kstart; k<kend; ++k)
        for (int j=jstart; j<jend; ++j)
            #pragma ivdep
            for (int i=istart; i<iend; ++i)
            {
                const int ijk = i + j*icells + k*ijcells;

                at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
                        +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi
                        +  visc * dg4(a[ijk-kk3], a[ijk-kk2], a[ijk-kk1], a[ijk], a[ijk+kk1], a[ijk+kk2], a[ijk+kk3])*dzidzi;
            }
}

/*
4th order diffusion (3D), no shared memory use, quite similar to GPU implementation MicroHH
*/
__global__ void diff_gpu_3d(double * const __restrict__ at, const double * const __restrict__ a,
        const double dxidxi, const double dyidyi, const double dzidzi,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart, const int kend,
        const int icells, const int ijcells)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const double visc = 0.1;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk = i + j*icells + k*ijcells;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*icells;
        const int jj2 = 2*icells;
        const int jj3 = 3*icells;
        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;

        at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
                +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi
                +  visc * dg4(a[ijk-kk3], a[ijk-kk2], a[ijk-kk1], a[ijk], a[ijk+kk1], a[ijk+kk2], a[ijk+kk3])*dzidzi;
    }
}

/*
4th order diffusion, shared memory for horizontal (i,j) stencil, global memory for vertical (k) stencil
*/
__global__ void diff_gpu_3d_s2d(double * const __restrict__ at, const double * const __restrict__ a,
        const double dxidxi, const double dyidyi, const double dzidzi,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart, const int kend,
        const int icells, const int ijcells, const int ngc)
{
    extern __shared__ double as[];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k  = blockIdx.z + kstart;
    const int blockxpad = blockDim.x+2*ngc;

    const double visc = 0.1;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk  = i + j*icells + k*ijcells;      // index in global memory
        const int ijks = (tx+ngc) + (ty+ngc)*blockxpad; // Same location in 2d shared mem

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj3 = 3*icells;

        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;

        const int jjs1 = 1*blockxpad;
        const int jjs2 = 2*blockxpad;
        const int jjs3 = 3*blockxpad;

        as[ijks] = a[ijk];

        if(ty < ngc)
            as[ijks-jjs3] = a[ijk-jj3];
        if(ty >= blockDim.y-ngc)
            as[ijks+jjs3] = a[ijk+jj3];

        if(tx < ngc)
            as[ijks-ii3] = a[ijk-ii3];
        if(tx >= blockDim.x-ngc)
            as[ijks+ii3] = a[ijk+ii3];

        __syncthreads();

        at[ijk] += visc * dg4(as[ijks-ii3 ], as[ijks-ii2 ], as[ijks-ii1 ], as[ijks], as[ijks+ii1 ], as[ijks+ii2 ], as[ijks+ii3 ])*dxidxi
                +  visc * dg4(as[ijks-jjs3], as[ijks-jjs2], as[ijks-jjs1], as[ijks], as[ijks+jjs1], as[ijks+jjs2], as[ijks+jjs3])*dyidyi
                +  visc * dg4(a [ijk-kk3  ], a [ijk-kk2  ], a [ijk-kk1  ], as[ijks], a [ijk+kk1  ], a [ijk+kk2  ], a [ijk+kk3  ])*dzidzi;
    }
}

/*
4th order diffusion, shared memory for horizontal (i,j) stencil, vertical stencil in local variables, vertical loop on GPU
*/
__global__ void diff_gpu_3d_s3d(double * const __restrict__ at, const double * const __restrict__ a,
        const double dxidxi, const double dyidyi, const double dzidzi,
        const int istart, const int iend,
        const int jstart, const int jend,
        const int kstart, const int kend,
        const int icells, const int ijcells, const int ngc)
{
    extern __shared__ double as[];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int blockxpad = blockDim.x+2*ngc;

    const double visc = 0.1;

    if(i < iend && j < jend)
    {
        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj3 = 3*icells;

        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;
        const int kk4 = 4*ijcells;

        const int jjs1 = 1*blockxpad;
        const int jjs2 = 2*blockxpad;
        const int jjs3 = 3*blockxpad;

        int ijk = i +j*icells + kstart*ijcells;
        double akm3, akm2, akm1, aijk, akp1, akp2, akp3;

        // Read vertical stencil into variables
        akm3 = a[ijk-kk3];
        akm2 = a[ijk-kk2];
        akm1 = a[ijk-kk1];
        aijk = a[ijk    ];
        akp1 = a[ijk+kk1];
        akp2 = a[ijk+kk2];
        akp3 = a[ijk+kk3];

        for (int k=kstart; k<kend; ++k)
        {
            const int ijk  = i + j*icells + k*ijcells;      // index in global memory
            const int ijks = (tx+ngc) + (ty+ngc)*blockxpad; // Same location in 2d shared mem

            __syncthreads();

            as[ijks] = aijk;

            if(ty < ngc)
                as[ijks-jjs3] = a[ijk-jj3];
            if(ty >= blockDim.y-ngc)
                as[ijks+jjs3] = a[ijk+jj3];

            if(tx < ngc)
                as[ijks-ii3] = a[ijk-ii3];
            if(tx >= blockDim.x-ngc)
                as[ijks+ii3] = a[ijk+ii3];

            __syncthreads();

            at[ijk] += visc * dg4(as[ijks-ii3 ], as[ijks-ii2 ], as[ijks-ii1 ], as[ijks], as[ijks+ii1 ], as[ijks+ii2 ], as[ijks+ii3 ])*dxidxi
                    +  visc * dg4(as[ijks-jjs3], as[ijks-jjs2], as[ijks-jjs1], as[ijks], as[ijks+jjs1], as[ijks+jjs2], as[ijks+jjs3])*dyidyi
                    +  visc * dg4(akm3,          akm2,          akm1,          aijk,     akp1,          akp2,          akp3         )*dzidzi;

            // Shift vertical stencil
            akm3 = akm2;
            akm2 = akm1;
            akm1 = aijk;
            aijk = akp1;
            akp1 = akp2;
            akp2 = akp3;
            if(k < kend-1)
                akp3 = a[ijk+kk4];
        }
    }
}


/*
Get max difference between two fields
*/
double maxdiff(const double * const __restrict__ a, const double * const __restrict__ b, const int n)
{
    double maxdiff=0;
    double diff=0;
    for(int i=0; i<n; ++i)
    {
        diff = std::abs(a[i]-b[i]);
        if(diff > maxdiff)
            maxdiff = diff;
    }
    return maxdiff;
}

int main()
{
    //
    // Grid
    //
    const double dxi = 0.1;
    const double dyi = 0.1;
    const double dzi = 0.1;

    const int itot = 256;
    const int jtot = 256;
    const int ktot = 256;
    const int gc   = 3;
    const int iter = 50;

    //
    // Calculate the required variables.
    //
    const int ncells  = (itot+2*gc)*(jtot+2*gc)*(ktot+2*gc);
    const int istart  = gc;
    const int jstart  = gc;
    const int kstart  = gc;
    const int iend    = itot+gc;
    const int jend    = jtot+gc;
    const int kend    = ktot+gc;
    const int icells  = itot+2*gc;
    const int ijcells = (itot+2*gc)*(jtot+2*gc);

    //
    // Prepare fields on HOST
    //
    double *a    = new double[ncells];
    double *at   = new double[ncells];
    double *tmp1 = new double[ncells];

    for (int n=0; n<ncells; ++n)
    {
    	a [n]   = 0.001 * (std::rand() % 1000) - 0.5;
    	at[n]   = 0.;
    	tmp1[n] = 0.;
    }

    //
    // Prepare fields on DEVICE
    //
    double *ad, *atd;
    hipMalloc((void **)&ad,  ncells*sizeof(double));
    hipMalloc((void **)&atd, ncells*sizeof(double));

    hipMemcpy(ad,  a,  ncells*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(atd, at, ncells*sizeof(double), hipMemcpyHostToDevice);

    //
    // CUDA thread blocks
    //
    const int blocki = 32;
    const int blockj = 8;
    const int gridi  = itot/blocki + (itot%blocki > 0);
    const int gridj  = jtot/blockj + (jtot%blockj > 0);
    dim3 gridGPU  (gridi, gridj, ktot);
    dim3 gridGPU2d(gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    //
    // Timer stuff
    //
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float  dt1, dt2;

    //
    // Execute kernels
    //
    //////////////////// CPU //////////////////////////
    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n)
    {
       diff_cpu_3d(at,  a,  dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt1, startEvent, stopEvent);
    printf("CPU; elapsed=%f [ms]\n",dt1);

    //////////////////// GPU //////////////////////////
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n)
    {
        diff_gpu_3d<<<gridGPU, blockGPU>>>
                 (atd, ad, dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells);

        //diff_gpu_3d_s2d<<<gridGPU, blockGPU, (blocki+2*gc)*(blockj+2*gc)*sizeof(double)>>>
        //         (atd, ad, dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells, gc);

        //diff_gpu_3d_s3d<<<gridGPU2d, blockGPU, (blocki+2*gc)*(blockj+2*gc)*sizeof(double)>>>
        //         (atd, ad, dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells, gc);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt2, startEvent, stopEvent);

    //
    // Copy device field to tmp1
    //
    hipMemcpy(tmp1, atd, ncells*sizeof(double), hipMemcpyDeviceToHost);

    printf("GPU; elapsed=%f [ms], speedup=%f, maxdiff=%e \n",dt2,dt1/dt2,maxdiff(at,tmp1,ncells));

    return 0;
}
