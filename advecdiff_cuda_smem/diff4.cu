
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <stdlib.h>
#include <cstdio>

__host__ __device__ inline double dg4(const double v1, const double v2, const double v3, const double v4, const double v5, const double v6, const double v7) 
{
    return (1./576.)*(v1+v7) + (-54./576.)*(v2+v6) + (783./576.)*(v3+v5) + (-1460./576.)*v4;
}

/* 
4th order diffusion (2D), similar to CPU implementation MicroHH
*/
void diff_cpu_2d(double * const __restrict__ at, const double * const __restrict__ a,
                 const double dxidxi, const double dyidyi, const double dzidzi,
                 const int istart, const int iend, 
                 const int jstart, const int jend, 
                 const int kstart, const int kend, 
                 const int icells, const int ijcells)
{
    const int ii1 = 1;
    const int ii2 = 2;
    const int ii3 = 3;
    const int jj1 = 1*icells;
    const int jj2 = 2*icells;
    const int jj3 = 3*icells;

    const double visc = 0.1;

    for (int k=kstart; k<kend; ++k)
        for (int j=jstart; j<jend; ++j)
            #pragma ivdep
            for (int i=istart; i<iend; ++i)
            {
                const int ijk = i + j*icells + k*ijcells;

                at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
                        +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi;
            }
}

/* 
4th order diffusion (3D), similar to CPU implementation MicroHH
*/
void diff_cpu_3d(double * const __restrict__ at, const double * const __restrict__ a,
                 const double dxidxi, const double dyidyi, const double dzidzi,
                 const int istart, const int iend, 
                 const int jstart, const int jend, 
                 const int kstart, const int kend, 
                 const int icells, const int ijcells)
{
    const int ii1 = 1;
    const int ii2 = 2;
    const int ii3 = 3;
    const int jj1 = 1*icells;
    const int jj2 = 2*icells;
    const int jj3 = 3*icells;
    const int kk1 = 1*ijcells;
    const int kk2 = 2*ijcells;
    const int kk3 = 3*ijcells;

    const double visc = 0.1;

    for (int k=kstart; k<kend; ++k)
        for (int j=jstart; j<jend; ++j)
            #pragma ivdep
            for (int i=istart; i<iend; ++i)
            {
                const int ijk = i + j*icells + k*ijcells;

                at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
                        +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi
                        +  visc * dg4(a[ijk-kk3], a[ijk-kk2], a[ijk-kk1], a[ijk], a[ijk+kk1], a[ijk+kk2], a[ijk+kk3])*dzidzi;
            }
}

/* 
4th order diffusion (2D), no shared memory use
*/
__global__ void diff_gpu_2d(double * const __restrict__ at, const double * const __restrict__ a,
                            const double dxidxi, const double dyidyi, const double dzidzi,
                            const int istart, const int iend, 
                            const int jstart, const int jend, 
                            const int kstart, const int kend, 
                            const int icells, const int ijcells)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const double visc = 0.1;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk = i + j*icells + k*ijcells;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*icells;
        const int jj2 = 2*icells;
        const int jj3 = 3*icells;

	at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
	        +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi;
    }
}

/* 
4th order diffusion (3D), no shared memory use
*/
__global__ void diff_gpu_3d(double * const __restrict__ at, const double * const __restrict__ a,
                            const double dxidxi, const double dyidyi, const double dzidzi,
                            const int istart, const int iend, 
                            const int jstart, const int jend, 
                            const int kstart, const int kend, 
                            const int icells, const int ijcells)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const double visc = 0.1;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk = i + j*icells + k*ijcells;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*icells;
        const int jj2 = 2*icells;
        const int jj3 = 3*icells;
        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;

	at[ijk] += visc * dg4(a[ijk-ii3], a[ijk-ii2], a[ijk-ii1], a[ijk], a[ijk+ii1], a[ijk+ii2], a[ijk+ii3])*dxidxi
	        +  visc * dg4(a[ijk-jj3], a[ijk-jj2], a[ijk-jj1], a[ijk], a[ijk+jj1], a[ijk+jj2], a[ijk+jj3])*dyidyi
	        +  visc * dg4(a[ijk-kk3], a[ijk-kk2], a[ijk-kk1], a[ijk], a[ijk+kk1], a[ijk+kk2], a[ijk+kk3])*dzidzi;
    }
}

/* 
4th order diffusion, 2D smem tile
*/
__global__ void diff_gpu_2d_s2d(double * const __restrict__ at, const double * const __restrict__ a,
                                const double dxidxi, const double dyidyi, const double dzidzi,
                                const int istart, const int iend, 
                                const int jstart, const int jend, 
                                const int kstart, const int kend, 
                                const int icells, const int ijcells, const int ngc)
{
    extern __shared__ double as[]; 

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k  = blockIdx.z + kstart;
    const int blockxpad = blockDim.x+2*ngc;

    const double visc = 0.1;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk  = i + j*icells + k*ijcells; // index in global memory
        const int ijks = (tx+ngc) + (ty+ngc)*blockxpad; // Same location in 2d shared mem

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj3 = 3*icells;

        const int jjs1 = 1*blockxpad;
        const int jjs2 = 2*blockxpad;
        const int jjs3 = 3*blockxpad;

        as[ijks] = a[ijk];

        if(ty < ngc)
            as[ijks-jjs3] = a[ijk-jj3];
        if(ty >= blockDim.y-ngc)
            as[ijks+jjs3] = a[ijk+jj3];

        if(tx < ngc)
            as[ijks-ii3] = a[ijk-ii3];
        if(tx >= blockDim.x-ngc)
            as[ijks+ii3] = a[ijk+ii3];

        __syncthreads();

	at[ijk] += visc * dg4(as[ijks-ii3 ], as[ijks-ii2 ], as[ijks-ii1 ], as[ijks], as[ijks+ii1 ], as[ijks+ii2 ], as[ijks+ii3 ])*dxidxi
	        +  visc * dg4(as[ijks-jjs3], as[ijks-jjs2], as[ijks-jjs1], as[ijks], as[ijks+jjs1], as[ijks+jjs2], as[ijks+jjs3])*dyidyi;
    }
}

/* 
4th order diffusion (3d), 2D smem tile
*/
//__global__ void diff_gpu_3d_s2d(double * const __restrict__ at, const double * const __restrict__ a,
//                                const double dxidxi, const double dyidyi, const double dzidzi,
//                                const int istart, const int iend, 
//                                const int jstart, const int jend, 
//                                const int kstart, const int kend, 
//                                const int icells, const int ijcells, const int ngc)
//{
//    extern __shared__ double as[]; 
//
//    const int tx = threadIdx.x;
//    const int ty = threadIdx.y;
//    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
//    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
//    const int k  = blockIdx.z + kstart;
//    const int blockxpad = blockDim.x+2*ngc;
//
//    const double visc = 0.1;
//
//    if(i < iend && j < jend && k < kend)
//    {
//        const int ijk  = i + j*icells + k*ijcells; // index in global memory
//        const int ijks = (tx+ngc) + (ty+ngc)*blockxpad; // Same location in 2d shared mem
//
//        const int ii1 = 1;
//        const int ii2 = 2;
//        const int ii3 = 3;
//        const int jj3 = 3*icells;
//        const int kk1 = 1*ijcells;
//        const int kk2 = 2*ijcells;
//        const int kk3 = 3*ijcells;
//
//        const int jjs1 = 1*blockxpad;
//        const int jjs2 = 2*blockxpad;
//        const int jjs3 = 3*blockxpad;
//
//        as[ijks] = a[ijk];
//
//        if(ty < ngc)
//            as[ijks-jjs3] = a[ijk-jj3];
//        if(ty >= blockDim.y-ngc)
//            as[ijks+jjs3] = a[ijk+jj3];
//
//        if(tx < ngc)
//            as[ijks-ii3] = a[ijk-ii3];
//        if(tx >= blockDim.x-ngc)
//            as[ijks+ii3] = a[ijk+ii3];
//
//        __syncthreads();
//
//	at[ijk] += visc * dg4(as[ijks-ii3 ], as[ijks-ii2 ], as[ijks-ii1 ], as[ijks], as[ijks+ii1 ], as[ijks+ii2 ], as[ijks+ii3 ])*dxidxi
//	        +  visc * dg4(as[ijks-jjs3], as[ijks-jjs2], as[ijks-jjs1], as[ijks], as[ijks+jjs1], as[ijks+jjs2], as[ijks+jjs3])*dyidyi
//	        +  visc * dg4(a [ijk-kk3],    a[ijk-kk2],   a [ijk-kk1],   as[ijks], a [ijk+kk1],   a [ijk+kk2],   a [ijk+kk3])*dzidzi;
//    }
//}

/* 
4th order diffusion (3d), 2D smem tile
*/
__global__ void diff_gpu_3d_s2d(double * const __restrict__ at, const double * const __restrict__ a,
                                const double dxidxi, const double dyidyi, const double dzidzi,
                                const int istart, const int iend, 
                                const int jstart, const int jend, 
                                const int kstart, const int kend, 
                                const int icells, const int ijcells, const int ngc)
{
    extern __shared__ double as[]; 

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    //const int k  = blockIdx.z + kstart;
    const int blockxpad = blockDim.x+2*ngc;

    const double visc = 0.1;

    if(i < iend && j < jend)
    {
        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj3 = 3*icells;
        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;

        const int jjs1 = 1*blockxpad;
        const int jjs2 = 2*blockxpad;
        const int jjs3 = 3*blockxpad;

        for(int k=kstart; k<kend; ++k)
        {
            const int ijk  = i + j*icells + k*ijcells; // index in global memory
            const int ijks = (tx+ngc) + (ty+ngc)*blockxpad; // Same location in 2d shared mem

            as[ijks] = a[ijk];

            if(ty < ngc)
                as[ijks-jjs3] = a[ijk-jj3];
            if(ty >= blockDim.y-ngc)
                as[ijks+jjs3] = a[ijk+jj3];

            if(tx < ngc)
                as[ijks-ii3] = a[ijk-ii3];
            if(tx >= blockDim.x-ngc)
                as[ijks+ii3] = a[ijk+ii3];

            __syncthreads();

	    at[ijk] += visc * dg4(as[ijks-ii3 ], as[ijks-ii2 ], as[ijks-ii1 ], as[ijks], as[ijks+ii1 ], as[ijks+ii2 ], as[ijks+ii3 ])*dxidxi
	            +  visc * dg4(as[ijks-jjs3], as[ijks-jjs2], as[ijks-jjs1], as[ijks], as[ijks+jjs1], as[ijks+jjs2], as[ijks+jjs3])*dyidyi
	            +  visc * dg4(a [ijk-kk3],    a[ijk-kk2],   a [ijk-kk1],   as[ijks], a [ijk+kk1],   a [ijk+kk2],   a [ijk+kk3])*dzidzi;
        }
    }
}


/* 
Get max difference between two fields
*/
double maxdiff(const double * const __restrict__ a, const double * const __restrict__ b, const int n)
{
    double maxdiff=0;
    double diff=0;
    for(int i=0; i<n; ++i)
    {
        diff = std::abs(a[i]-b[i]);
        if(diff > maxdiff)
            maxdiff = diff;
    }
    return maxdiff;
}

int main()
{
    //
    // Grid
    //
    const double dxi = 0.1;
    const double dyi = 0.1;
    const double dzi = 0.1;
    
    const int itot = 256;
    const int jtot = 256;
    const int ktot = 256;
    const int gc   = 3;
    const int iter = 40;
    
    //
    // Calculate the required variables.
    //
    const int ncells  = (itot+2*gc)*(jtot+2*gc)*(ktot+2*gc);
    const int istart  = gc;
    const int jstart  = gc;
    const int kstart  = gc;
    const int iend    = itot+gc;
    const int jend    = jtot+gc;
    const int kend    = ktot+gc;
    const int icells  = itot+2*gc;
    const int jcells  = jtot+2*gc;
    const int kcells  = ktot+2*gc;
    const int ijcells = (itot+2*gc)*(jtot+2*gc);
    
    // Padded settings, interior aligned to 128 byte blocks
    const int mo        = 16 - gc;           // Padding at start of array 
    const int pl        = 16-(int)itot%16;   // Elements left in last 128 byte block
    const int icellsp   = itot + pl + (pl < 2*gc)*16;
    const int ijcellsp  = icellsp * jcells;  
    const int ncellsp   = ijcellsp * kcells + mo;
    
    //
    // Prepare fields on HOST
    //
    double *a    = new double[ncells];
    double *at   = new double[ncells];
    double *tmp1 = new double[ncells];
    
    for (int n=0; n<ncells; ++n)
    {
    	a [n]   = 0.001 * (std::rand() % 1000) - 0.5;
    	at[n]   = 0.;
    	tmp1[n] = 0.;
    }
    
    // 
    // Prepare fields on DEVICE
    //
    double *ad, *atd;
    hipMalloc((void **)&ad,  ncellsp*sizeof(double));
    hipMalloc((void **)&atd, ncellsp*sizeof(double));
    hipMemcpy2D(&ad[mo],  icellsp*sizeof(double),  a,  icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&atd[mo], icellsp*sizeof(double),  at, icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    
    //
    // CUDA thread blocks
    //
    const int blocki = 32;
    const int blockj = 16;
    const int gridi  = itot/blocki + (itot%blocki > 0);
    const int gridj  = jtot/blockj + (jtot%blockj > 0);
    dim3 gridGPU  (gridi, gridj, ktot);
    dim3 gridGPU2d(gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    // 
    // Timer stuff
    //
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float  dt1, dt2;

    //
    // Execute kernels
    //
    //////////////////// CPU //////////////////////////
    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n) // iter+1 since GPU version is warmed up with one call
    {
       //diff_cpu_2d(at,  a,  dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells);
       diff_cpu_3d(at,  a,  dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icells, ijcells);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt1, startEvent, stopEvent);
    printf("CPU; elapsed=%f [ms]\n",dt1);
 
    //////////////////// GPU //////////////////////////
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n)
    {
        //diff_gpu_2d<<<gridGPU, blockGPU>>> 
        //         (&atd[mo], &ad[mo], dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp);

        //diff_gpu_3d<<<gridGPU, blockGPU>>> 
        //         (&atd[mo], &ad[mo], dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp);

        //diff_gpu_2d_s2d<<<gridGPU, blockGPU, (blocki+2*gc)*(blockj+2*gc)*sizeof(double)>>> 
        //         (&atd[mo], &ad[mo], dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp, gc);

        diff_gpu_3d_s2d<<<gridGPU2d, blockGPU, (blocki+2*gc)*(blockj+2*gc)*sizeof(double)>>> 
                 (&atd[mo], &ad[mo], dxi, dyi, dzi, istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp, gc);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt2, startEvent, stopEvent);

    //
    // Copy device field to tmp1 
    //
    hipMemcpy2D(tmp1, icells*sizeof(double), &atd[mo], icellsp*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyDeviceToHost);

    printf("GPU; elapsed=%f [ms], speedup=%f, maxdiff=%e \n",dt2,dt1/dt2,maxdiff(at,tmp1,ncells));

    return 0;
}
