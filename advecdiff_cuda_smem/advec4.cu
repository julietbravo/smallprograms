
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <stdlib.h>
#include <cstdio>

// Fourth order interpolation function.
__host__ __device__ inline double interp(const double m2, const double m1, const double p1, const double p2)
{
  return (-1./16)*(m2+p2) + (9./16)*(m1+p1);
}

// Fourth order gradient function.
__host__ __device__ inline double grad(const double m2, const double m1, const double p1, const double p2)
{
  return (1./24.)*(m2-p2) + (27./24.)*(p1-m1);
}

/* 
4th order advection on cpu
*/
void advec_cpu(double * const __restrict__ ut, 
               const double * const __restrict__ u, const double * const __restrict__ v, const double * const __restrict__ w,
               const int istart, const int iend, 
               const int jstart, const int jend, 
               const int kstart, const int kend, 
               const int icells, const int ijcells)
{
    const int ii1 = 1;
    const int ii2 = 2;
    const int ii3 = 3;
    const int jj1 = 1*icells;
    const int jj2 = 2*icells;
    const int jj3 = 3*icells;
    const int kk1 = 1*ijcells;
    const int kk2 = 2*ijcells;
    const int kk3 = 3*ijcells;

    for (int k=kstart; k<kend; ++k)
        for (int j=jstart; j<jend; ++j)
            #pragma ivdep
            for (int i=istart; i<iend; ++i)
            {
                const int ijk = i + j*icells + k*ijcells;

                ut[ijk] += grad( interp( u[ijk-ii3], u[ijk-ii2], u[ijk-ii1], u[ijk    ] ) * interp( u[ijk-ii3], u[ijk-ii2], u[ijk-ii1], u[ijk    ] ),
                                 interp( u[ijk-ii2], u[ijk-ii1], u[ijk    ], u[ijk+ii1] ) * interp( u[ijk-ii2], u[ijk-ii1], u[ijk    ], u[ijk+ii1] ),
                                 interp( u[ijk-ii1], u[ijk    ], u[ijk+ii1], u[ijk+ii2] ) * interp( u[ijk-ii1], u[ijk    ], u[ijk+ii1], u[ijk+ii2] ),
                                 interp( u[ijk    ], u[ijk+ii1], u[ijk+ii2], u[ijk+ii3] ) * interp( u[ijk    ], u[ijk+ii1], u[ijk+ii2], u[ijk+ii3] ))

                         + grad( interp( v[ijk-ii2-jj1], v[ijk-ii1-jj1], v[ijk-jj1], v[ijk+ii1-jj1] ) * interp( u[ijk-jj3], u[ijk-jj2], u[ijk-jj1], u[ijk    ] ),
                                 interp( v[ijk-ii2    ], v[ijk-ii1    ], v[ijk    ], v[ijk+ii1    ] ) * interp( u[ijk-jj2], u[ijk-jj1], u[ijk    ], u[ijk+jj1] ),
                                 interp( v[ijk-ii2+jj1], v[ijk-ii1+jj1], v[ijk+jj1], v[ijk+ii1+jj1] ) * interp( u[ijk-jj1], u[ijk    ], u[ijk+jj1], u[ijk+jj2] ),
                                 interp( v[ijk-ii2+jj2], v[ijk-ii1+jj2], v[ijk+jj2], v[ijk+ii1+jj2] ) * interp( u[ijk    ], u[ijk+jj1], u[ijk+jj2], u[ijk+jj3] ))

                         + grad( interp( w[ijk-ii2-kk1], w[ijk-ii1-kk1], w[ijk-kk1], w[ijk+ii1-kk1] ) * interp( u[ijk-kk3], u[ijk-kk2], u[ijk-kk1], u[ijk    ] ),
                                 interp( w[ijk-ii2    ], w[ijk-ii1    ], w[ijk    ], w[ijk+ii1    ] ) * interp( u[ijk-kk2], u[ijk-kk1], u[ijk    ], u[ijk+kk1] ),
                                 interp( w[ijk-ii2+kk1], w[ijk-ii1+kk1], w[ijk+kk1], w[ijk+ii1+kk1] ) * interp( u[ijk-kk1], u[ijk    ], u[ijk+kk1], u[ijk+kk2] ),
                                 interp( w[ijk-ii2+kk2], w[ijk-ii1+kk2], w[ijk+kk2], w[ijk+ii1+kk2] ) * interp( u[ijk    ], u[ijk+kk1], u[ijk+kk2], u[ijk+kk3] ));
            }
}


/* 
4th order advection (3D), no shared memory use
*/
__global__ void advec_gpu(double * const __restrict__ ut, 
                          const double * const __restrict__ u, const double * const __restrict__ v, const double * const __restrict__ w,
                          const int istart, const int iend, 
                          const int jstart, const int jend, 
                          const int kstart, const int kend, 
                          const int icells, const int ijcells)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    if(i < iend && j < jend && k < kend)
    {
        const int ijk = i + j*icells + k*ijcells;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*icells;
        const int jj2 = 2*icells;
        const int jj3 = 3*icells;
        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;

        ut[ijk] += grad( interp( u[ijk-ii3], u[ijk-ii2], u[ijk-ii1], u[ijk    ] ) * interp( u[ijk-ii3], u[ijk-ii2], u[ijk-ii1], u[ijk    ] ),
                         interp( u[ijk-ii2], u[ijk-ii1], u[ijk    ], u[ijk+ii1] ) * interp( u[ijk-ii2], u[ijk-ii1], u[ijk    ], u[ijk+ii1] ),
                         interp( u[ijk-ii1], u[ijk    ], u[ijk+ii1], u[ijk+ii2] ) * interp( u[ijk-ii1], u[ijk    ], u[ijk+ii1], u[ijk+ii2] ),
                         interp( u[ijk    ], u[ijk+ii1], u[ijk+ii2], u[ijk+ii3] ) * interp( u[ijk    ], u[ijk+ii1], u[ijk+ii2], u[ijk+ii3] ))

                 + grad( interp( v[ijk-ii2-jj1], v[ijk-ii1-jj1], v[ijk-jj1], v[ijk+ii1-jj1] ) * interp( u[ijk-jj3], u[ijk-jj2], u[ijk-jj1], u[ijk    ] ),
                         interp( v[ijk-ii2    ], v[ijk-ii1    ], v[ijk    ], v[ijk+ii1    ] ) * interp( u[ijk-jj2], u[ijk-jj1], u[ijk    ], u[ijk+jj1] ),
                         interp( v[ijk-ii2+jj1], v[ijk-ii1+jj1], v[ijk+jj1], v[ijk+ii1+jj1] ) * interp( u[ijk-jj1], u[ijk    ], u[ijk+jj1], u[ijk+jj2] ),
                         interp( v[ijk-ii2+jj2], v[ijk-ii1+jj2], v[ijk+jj2], v[ijk+ii1+jj2] ) * interp( u[ijk    ], u[ijk+jj1], u[ijk+jj2], u[ijk+jj3] ))

                 + grad( interp( w[ijk-ii2-kk1], w[ijk-ii1-kk1], w[ijk-kk1], w[ijk+ii1-kk1] ) * interp( u[ijk-kk3], u[ijk-kk2], u[ijk-kk1], u[ijk    ] ),
                         interp( w[ijk-ii2    ], w[ijk-ii1    ], w[ijk    ], w[ijk+ii1    ] ) * interp( u[ijk-kk2], u[ijk-kk1], u[ijk    ], u[ijk+kk1] ),
                         interp( w[ijk-ii2+kk1], w[ijk-ii1+kk1], w[ijk+kk1], w[ijk+ii1+kk1] ) * interp( u[ijk-kk1], u[ijk    ], u[ijk+kk1], u[ijk+kk2] ),
                         interp( w[ijk-ii2+kk2], w[ijk-ii1+kk2], w[ijk+kk2], w[ijk+ii1+kk2] ) * interp( u[ijk    ], u[ijk+kk1], u[ijk+kk2], u[ijk+kk3] ));
    }
}

/* 
4th order advection, smem
*/
__global__ void advec_gpu_smem(double * const __restrict__ ut, 
                               const double * const __restrict__ u, const double * const __restrict__ v, const double * const __restrict__ w, 
                               const int istart, const int iend, 
                               const int jstart, const int jend, 
                               const int kstart, const int kend, 
                               const int icells, const int ijcells, const int ngc)
{
    extern __shared__ double shared[]; 
    const int smem_block = (blockDim.x + 2*ngc) * (blockDim.y + 2*ngc);
    double *us = &shared[0];
    double *vs = &shared[smem_block];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int blockxpad = blockDim.x+2*ngc;

    if(i < iend && j < jend)
    {
        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;
        const int jj1 = 1*icells;
        const int jj2 = 2*icells;
        const int jj3 = 3*icells;
        const int kk1 = 1*ijcells;
        const int kk2 = 2*ijcells;
        const int kk3 = 3*ijcells;
        const int kk4 = 4*ijcells;

        const int jjs1 = 1*blockxpad;
        const int jjs2 = 2*blockxpad;
        const int jjs3 = 3*blockxpad;

        int ijk;
        const int ijks = (tx+ngc) + (ty+ngc)*blockxpad;

        for(int k=kstart; k<kend; ++k)
        {
            ijk  = i + j*icells + k*ijcells; // index in global memory
 
            us[ijks] = u[ijk];
            vs[ijks] = v[ijk];

            if(ty < ngc)
            {
                us[ijks-jjs3] = u[ijk-jj3];
                vs[ijks-jjs3] = v[ijk-jj3];
            }
            if(ty >= blockDim.y-ngc)
            {
                us[ijks+jjs3] = u[ijk+jj3];
                vs[ijks+jjs3] = v[ijk+jj3];
            }

            if(tx < ngc)
            {
                us[ijks-ngc] = u[ijk-ngc];
                vs[ijks-ngc] = v[ijk-ngc];

                if(ty < ngc)
                {
                    us[ijks-jjs3-ngc] = u[ijk-jj3-ngc];
                    vs[ijks-jjs3-ngc] = v[ijk-jj3-ngc];
                }
                if(ty >= blockDim.y-ngc)
                {
                    us[ijks+jjs3-ngc] = u[ijk+jj3-ngc];
                    vs[ijks+jjs3-ngc] = v[ijk+jj3-ngc];
                }
            }
            if(tx >= blockDim.x-ngc)
            {
                us[ijks+ngc] = u[ijk+ngc];
                vs[ijks+ngc] = v[ijk+ngc];

                if(ty < ngc)
                {
                    us[ijks-jjs3+ngc] = u[ijk-jj3+ngc];
                    vs[ijks-jjs3+ngc] = v[ijk-jj3+ngc];
                }
                if(ty >= blockDim.y-ngc)
                {
                    us[ijks+jjs3+ngc] = u[ijk+jj3+ngc];
                    vs[ijks+jjs3+ngc] = v[ijk+jj3+ngc];
                }
            }

            __syncthreads();

            ut[ijk] += grad( interp( us[ijks-ii3], us[ijks-ii2], us[ijks-ii1], us[ijks    ] ) * interp( us[ijks-ii3], us[ijks-ii2], us[ijks-ii1], us[ijks    ] ),
                             interp( us[ijks-ii2], us[ijks-ii1], us[ijks    ], us[ijks+ii1] ) * interp( us[ijks-ii2], us[ijks-ii1], us[ijks    ], us[ijks+ii1] ),
                             interp( us[ijks-ii1], us[ijks    ], us[ijks+ii1], us[ijks+ii2] ) * interp( us[ijks-ii1], us[ijks    ], us[ijks+ii1], us[ijks+ii2] ),
                             interp( us[ijks    ], us[ijks+ii1], us[ijks+ii2], us[ijks+ii3] ) * interp( us[ijks    ], us[ijks+ii1], us[ijks+ii2], us[ijks+ii3] ))

                     + grad( interp( vs[ijks-ii2-jjs1], vs[ijks-ii1-jjs1], vs[ijks-jjs1], vs[ijks+ii1-jjs1] ) * interp( us[ijks-jjs3], us[ijks-jjs2], us[ijks-jjs1], us[ijks     ] ),
                             interp( vs[ijks-ii2    ],  vs[ijks-ii1     ], vs[ijks    ],  vs[ijks+ii1     ] ) * interp( us[ijks-jjs2], us[ijks-jjs1], us[ijks     ], us[ijks+jjs1] ),
                             interp( vs[ijks-ii2+jjs1], vs[ijks-ii1+jjs1], vs[ijks+jjs1], vs[ijks+ii1+jjs1] ) * interp( us[ijks-jjs1], us[ijks     ], us[ijks+jjs1], us[ijks+jjs2] ),
                             interp( vs[ijks-ii2+jjs2], vs[ijks-ii1+jjs2], vs[ijks+jjs2], vs[ijks+ii1+jjs2] ) * interp( us[ijks     ], us[ijks+jjs1], us[ijks+jjs2], us[ijks+jjs3] ))

                     + grad( interp( w[ijk-ii2-kk1], w[ijk-ii1-kk1], w[ijk-kk1], w[ijk+ii1-kk1] ) * interp( u[ijk-kk3], u[ijk-kk2], u[ijk-kk1], us[ijks  ] ),
                             interp( w[ijk-ii2    ], w[ijk-ii1    ], w[ijk    ], w[ijk+ii1    ] ) * interp( u[ijk-kk2], u[ijk-kk1], us[ijks  ], u[ijk+kk1] ),
                             interp( w[ijk-ii2+kk1], w[ijk-ii1+kk1], w[ijk+kk1], w[ijk+ii1+kk1] ) * interp( u[ijk-kk1], us[ijks  ], u[ijk+kk1], u[ijk+kk2] ),
                             interp( w[ijk-ii2+kk2], w[ijk-ii1+kk2], w[ijk+kk2], w[ijk+ii1+kk2] ) * interp( us[ijks  ], u[ijk+kk1], u[ijk+kk2], u[ijk+kk3] ));
        }
    }
}


/* 
Get max difference between two fields
*/
double maxdiff(const double * const __restrict__ a, const double * const __restrict__ b, const int n)
{
    double maxdiff=0;
    double diff=0;
    for(int i=0; i<n; ++i)
    {
        diff = std::abs(a[i]-b[i]);
        if(diff > maxdiff)
            maxdiff = diff;
    }
    return maxdiff;
}

int main()
{
    //
    // Grid
    //
    const int itot = 256;
    const int jtot = 256;
    const int ktot = 256;
    const int gc   = 3;
    const int iter = 10;
    
    //
    // Calculate the required variables.
    //
    const int ncells  = (itot+2*gc)*(jtot+2*gc)*(ktot+2*gc);
    const int istart  = gc;
    const int jstart  = gc;
    const int kstart  = gc;
    const int iend    = itot+gc;
    const int jend    = jtot+gc;
    const int kend    = ktot+gc;
    const int icells  = itot+2*gc;
    const int jcells  = jtot+2*gc;
    const int kcells  = ktot+2*gc;
    const int ijcells = (itot+2*gc)*(jtot+2*gc);
    
    // Padded settings, interior aligned to 128 byte blocks
    const int mo        = 16 - gc;           // Padding at start of array 
    const int pl        = 16-(int)itot%16;   // Elements left in last 128 byte block
    const int icellsp   = itot + pl + (pl < 2*gc)*16;
    const int ijcellsp  = icellsp * jcells;  
    const int ncellsp   = ijcellsp * kcells + mo;
    
    //
    // Prepare fields on HOST
    //
    double *u    = new double[ncells];
    double *v    = new double[ncells];
    double *w    = new double[ncells];
    double *ut   = new double[ncells];
    double *tmp1 = new double[ncells];
    
    for (int n=0; n<ncells; ++n)
    {
    	u [n]   = 0.001 * (std::rand() % 1000) - 0.5;
    	v [n]   = 0.001 * (std::rand() % 1000) - 0.5;
    	w [n]   = 0.001 * (std::rand() % 1000) - 0.5;
    	ut[n]   = 0.;
    	tmp1[n] = 0.;
    }
    
    // 
    // Prepare fields on DEVICE
    //
    double *ud, *vd, *wd, *utd;
    hipMalloc((void **)&ud,  ncellsp*sizeof(double));
    hipMalloc((void **)&vd,  ncellsp*sizeof(double));
    hipMalloc((void **)&wd,  ncellsp*sizeof(double));
    hipMalloc((void **)&utd, ncellsp*sizeof(double));
    hipMemcpy2D(&ud[mo],  icellsp*sizeof(double),  u,  icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&vd[mo],  icellsp*sizeof(double),  v,  icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&wd[mo],  icellsp*sizeof(double),  w,  icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&utd[mo], icellsp*sizeof(double),  ut, icells*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyHostToDevice);
    
    //
    // CUDA thread blocks
    //
    const int blocki = 32;
    const int blockj = 8;
    const int gridi  = itot/blocki + (itot%blocki > 0);
    const int gridj  = jtot/blockj + (jtot%blockj > 0);
    dim3 gridGPU  (gridi, gridj, ktot);
    dim3 gridGPU2d(gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    // 
    // Timer stuff
    //
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float  dt1, dt2;

    //
    // Execute kernels
    //
    //////////////////// CPU //////////////////////////
    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n) // iter+1 since GPU version is warmed up with one call
    {
       advec_cpu(ut, u, v, w,istart, iend, jstart, jend, kstart, kend, icells, ijcells);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt1, startEvent, stopEvent);
    printf("CPU; elapsed=%f [ms]\n",dt1);
 
    ////////////////////// GPU //////////////////////////
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    hipEventRecord(startEvent, 0);
    for(int n=0; n<iter; ++n)
    {
        //advec_gpu<<<gridGPU, blockGPU>>> 
        //         (&utd[mo], &ud[mo], &vd[mo], &wd[mo], istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp);
        advec_gpu_smem<<<gridGPU2d, blockGPU, 2*(blocki+2*gc)*(blockj+2*gc)*sizeof(double)>>> 
                 (&utd[mo], &ud[mo], &vd[mo], &wd[mo], istart, iend, jstart, jend, kstart, kend, icellsp, ijcellsp, gc);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dt2, startEvent, stopEvent);

    //
    // Copy device field to tmp1 
    //
    hipMemcpy2D(tmp1, icells*sizeof(double), &utd[mo], icellsp*sizeof(double), icells*sizeof(double), jcells*kcells, hipMemcpyDeviceToHost);

    printf("GPU; elapsed=%f [ms], speedup=%f, maxdiff=%e \n",dt2,dt1/dt2,maxdiff(ut,tmp1,ncells));

    return 0;
}
