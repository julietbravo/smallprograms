#include <vector>
#include <iostream>
#include "stdio.h"
#include <hipfft/hipfft.h>

#define cuda_safe_call(err) __cuda_safe_call(err, __FILE__, __LINE__)
inline void __cuda_safe_call(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
        printf("cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
}

inline int check_cufft(hipfftResult err)
{
    if (err == HIPFFT_SUCCESS)
    {
        //std::cout << "cufft success!" << std::endl;
        return 0;
    }
    else
    {
        if (err == HIPFFT_INVALID_PLAN)
            printf("cuFFT plan error: INVALID PLAN\n");
        else if (err == HIPFFT_ALLOC_FAILED)
            printf("cuFFT plan error: ALLOC FAILED\n");
        else if (err == HIPFFT_INVALID_TYPE)
            printf("cuFFT plan error: INVALID TYPE\n");
        else if (err == HIPFFT_INVALID_VALUE)
            printf("cuFFT plan error: INVALID VALUE\n");
        else if (err == HIPFFT_INTERNAL_ERROR)
            printf("cuFFT plan error: INTERNAL ERROR\n");
        else if (err == HIPFFT_EXEC_FAILED)
            printf("cuFFT plan error: EXEC FAILED\n");
        else if (err == HIPFFT_SETUP_FAILED)
            printf("cuFFT plan error: SETUP FAILED\n");
        else if (err == HIPFFT_INVALID_SIZE)
            printf("cuFFT plan error: INVALID SIZE\n");
        else if (err == HIPFFT_UNALIGNED_DATA)
            printf("cuFFT plan error: UNALIGNED DATA\n");
        else 
            printf("cuFFT plan error: OTHER\n");

        return 1; 
    }
}

int main()
{
    // Grid et al.
    // ------------------------
    const int nloops = 1000;
    const int itot = 512;
    const int jtot = 512;
    const int ktot = 512;
    const int ncells = itot*jtot*ktot;

    // Field at host
    // ------------------------
    std::vector<double> field(ncells);

    // Create device field & tmp
    // ------------------------
    double* field_g;
    double* tmp_g;
    cuda_safe_call(hipMalloc((void**)&field_g, ncells*sizeof(double)));
    cuda_safe_call(hipMalloc((void**)&tmp_g, ncells*sizeof(double)));
    cuda_safe_call(hipMemcpy(field_g, field.data(), ncells, hipMemcpyHostToDevice));

    // Create FFT plan
    // ------------------------
    hipfftHandle iplanf;
    const int rank = 1;

    // Double input
    int i_ni[]    = {itot};
    int i_istride = 1;
    int i_idist   = itot;

    // Double-complex output
    int o_ni[]    = {itot/2+1};
    int o_istride = 1;
    int o_idist   = itot/2+1;

    check_cufft( hipfftPlanMany(&iplanf, rank, i_ni, i_ni, i_istride, i_idist, o_ni, o_istride, o_idist, HIPFFT_D2Z, jtot*ktot) ); 

    // Calculate FFTs
    // ------------------------
    for (int i=0; i<nloops; ++i)
    {
        check_cufft( hipfftExecD2Z(iplanf, (hipfftDoubleReal*)field_g, (hipfftDoubleComplex*)tmp_g) );
        hipDeviceSynchronize();
    }

    return 0;
}
